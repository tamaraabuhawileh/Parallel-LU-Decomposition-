#include "hip/hip_runtime.h"
%%writefile snippet2.cu
#include <stdlib.h>
#include <stdio.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <time.h>
using namespace std;




//LU Decompistion


__global__ void lower_for (float** a ,float** l,float** u, int size ,int i )
{
    // Lower triangular matrix
int k = blockIdx.x * blockDim.x + (threadIdx.x + i);


if (k < size)
        {
            if (i == k)
            {
                l[i][i] = 1.0; // Diagonal elements of L are 1
            }
            
            else
            {
                float sum = 0.0;
                for (int j = 0; j < i; j++)
                {
                    sum += l[k][j] * u[j][i];
                }
                l[k][i] = (a[k][i] - sum) / u[i][i];
            }
        }

}

__global__ void upper_for (float** a ,float** l,float** u, int size ,int i )
{
    // Upper triangular matrix
int k = blockIdx.x * blockDim.x + (threadIdx.x + i) ;


if (k < size)
        {
          float sum = 0.0;
            for (int j = 0; j < i; j++)
            {
                sum += l[i][j] * u[j][k];
            }
            u[i][k] = a[i][k] - sum;

        }

}






//print the matrix out
void print_matrix(float** matrix, int size)
{
    //for each row...
    for (int i = 0; i < size; i++)
    {
        //for each column
        for (int j = 0; j < size; j++)
        {
            //print out the cell
            cout << left << setw(9) << setprecision(4) << matrix[i][j] << left <<  setw(10);
        }
        //new line when ever row is done
        cout << endl;
    }
}





//fill the array with random values (done for a)
void random_fill(float** matrix, int size)
{
    //fill a with random values
    cout << "Producing random values " << endl;
    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            matrix[i][j] = ((rand()%10)+1) ;
        }
    }

    //Ensure the matrix is diagonal dominant to guarantee invertible-ness
    //diagCount well help keep track of which column the diagonal is in
    int diagCount = 0;
    float sum = 0;
    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            //Sum all column vaalues
            sum += abs(matrix[i][j]);
        }
        //Remove the diagonal  value from the sum
        sum -= abs(matrix[i][diagCount]);
        //Add a random value to the sum and place in diagonal position
        matrix[i][diagCount] = sum + ((rand()%5)+1);
        ++diagCount;
        sum = 0;
    }
}
  //allocate vectors on unified memory **

 void initialize_matrices(float** a, float** l, float** u, int size)
  {



  for(int i=0; i<size; i++){
    hipMallocManaged(&a[i], size * sizeof(float));
    hipMallocManaged(&l[i], size * sizeof(float));
    hipMallocManaged(&u[i], size * sizeof(float));

  }
}




int main(){
  int n=5;

  float **a, **l, **u;
   int i =0;

  srand(1);
  hipMallocManaged(&a, n * sizeof(float*));
  hipMallocManaged(&l, n * sizeof(float*));
  hipMallocManaged(&u, n * sizeof(float*));
initialize_matrices(a,l,u,n);
random_fill(a, n);


// add the clock
 double runtime;
  runtime = clock()/(double)CLOCKS_PER_SEC;
for( i=0; i<n; ++i){

    lower_for <<< 16,250 >>> (a , l, u, n , i );
    hipDeviceSynchronize();
    upper_for <<< 16,250 >>> ( a , l, u, n , i );
    hipDeviceSynchronize();

   }

runtime = clock() - runtime;
cout << "Runtime for LU Decomposition is: " << (runtime)/(double)(CLOCKS_PER_SEC) << endl;







// print the matrices
cout << "A Matrix: " << endl;
print_matrix(a, n);
cout << "L Matrix: " << endl;
print_matrix(l, n);

cout << "U Matrix: " << endl;
print_matrix(u, n);

cout << "Runtime for LU Decomposition is: " << (runtime)/float(CLOCKS_PER_SEC) << endl;


for(i=0; i<n; i++){
    hipFree(a[i]);
     hipFree(l[i]);
     hipFree(u[i]);
  }
   hipFree(a);
     hipFree(l);
     hipFree(u);

  return 0;
}